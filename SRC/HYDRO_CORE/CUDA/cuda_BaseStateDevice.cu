/* FastEddy®: SRC/HYDRO_CORE/CUDA/cuda_BaseStateDevice.cu 
* ©2016 University Corporation for Atmospheric Research
* 
* This file is licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
* http://www.apache.org/licenses/LICENSE-2.0
* 
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <float.h>
#include <math.h>
#include <fempi.h>
#include <grid.h>
#include <mem_utils.h>
#include <hydro_core.h>
#include <fecuda_Device_cu.h>
#include <cuda_gridDevice_cu.h>
#include <cuda_hydroCoreDevice_cu.h>

#include "cuda_advectionDevice.cu" 
#include "cuda_buoyancyDevice.cu" 
#include "cuda_coriolisDevice.cu" 
#include "cuda_pressureDevice.cu" 
#include "cuda_BCsDevice.cu"
#include "cuda_rayleighDampingDevice.cu" 
#include "cuda_surfaceLayerDevice.cu"
#include "cuda_sgsTurbDevice.cu"
#include "cuda_molecularDiffDevice.cu" 
#include "cuda_sgstkeDevice.cu" 
#include "cuda_largeScaleForcingsDevice.cu" 
#include "cuda_moistureDevice.cu" 
#include "cuda_filtersDevice.cu" 


/*##############------------------- BASESTATE submodule variable declarations ---------------------#################*/
/*---BASESTATE*/
float *hydroBaseStateFlds_d;   /*Base Adress of memory containing all prognostic variable fields base-states */
float *hydroBaseStatePres_d;   /*Base Adress of memory containing the diagnostic base-state pressure field */

/*#################------------ BASESTATE submodule function definitions ------------------#############*/
/*----->>>>> int cuda_BaseStateDeviceSetup();       ---------------------------------------------------------
 * Used to hipMalloc and hipMemcpy parameters and coordinate arrays, and for the BASESTATE_CUDA submodule.
*/
extern "C" int cuda_BaseStateDeviceSetup(){
   int errorCode = CUDA_BASESTATE_SUCCESS;
   int Nelems;

   /*Set the full memory block number of elements for base-state fields*/
   Nelems = (Nxp+2*Nh)*(Nyp+2*Nh)*(Nzp+2*Nh);
   /* Allocate the Base State arrays on the device */
   fecuda_DeviceMalloc(Nelems*2*sizeof(float), &hydroBaseStateFlds_d);  //Only rho and theta base-state variables
   fecuda_DeviceMalloc(Nelems*sizeof(float), &hydroBaseStatePres_d);  //Only base-state pressure 

   /* Send the Base State arrays down to the device */
   hipMemcpy(hydroBaseStateFlds_d, hydroBaseStateFlds, Nelems*2*sizeof(float), hipMemcpyHostToDevice);
   gpuErrchk( hipPeekAtLastError() ); /*Check for errors in the cudaMemCpy calls*/
   hipMemcpy(hydroBaseStatePres_d, hydroBaseStatePres, Nelems*sizeof(float), hipMemcpyHostToDevice); 
   gpuErrchk( hipPeekAtLastError() ); /*Check for errors in the cudaMemCpy calls*/

   return(errorCode);
} //end cuda_BaseStateDeviceSetup()

/*----->>>>> extern "C" int cuda_BaseStateDeviceCleanup();  -----------------------------------------------------------
Used to free all malloced memory by the BASESTATE submodule.
*/

extern "C" int cuda_BaseStateDeviceCleanup(){
   int errorCode = CUDA_BASESTATE_SUCCESS;

   /* Free any BASESTATE submodule arrays */
   hipFree(hydroBaseStateFlds_d);
   hipFree(hydroBaseStatePres_d);
 
   return(errorCode);

}//end cuda_BaseStateDeviceCleanup()
